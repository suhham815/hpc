#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "defs.h"

extern "C" void MY_MMult(int m, int n, int k, double *a, int lda,
              double *b, int ldb,
              double *c, int ldc) 
{
    double *d_a, *d_b, *d_c; 
    hipError_t err;

    err = hipMalloc((void**)&d_a, lda * m * sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error allocating d_a: %s\n", hipGetErrorString(err));
        return;
    }
    err = hipMalloc((void**)&d_b, ldb * k * sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error allocating d_a: %s\n", hipGetErrorString(err));
        return;
    }
    err = hipMalloc((void**)&d_c, ldc * m * sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error allocating d_a: %s\n", hipGetErrorString(err));
        return;
    }

    hipMemcpy(d_a, a, lda * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, ldb * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, ldb * n * sizeof(double), hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    double alpha = 1.0;
    double beta = 1.0;

    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                m, n, k, 
                &alpha, 
                d_a, lda, 
                d_b, ldb, 
                &beta, 
                d_c, ldc);

    hipMemcpy(c, d_c, ldc * n * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipblasDestroy(handle);
}